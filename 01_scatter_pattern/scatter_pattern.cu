#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

__host__ __device__ int outInvariant(int inValue) {
  return inValue * inValue;
}
__host__ __device__ int outDependent(int value, int inIdx, int outIdx) {
  if (inIdx == outIdx) {
    return 2 * value;
  } else if (inIdx > outIdx) {
    return value / (inIdx - outIdx);
  } else {
    return value / (outIdx - inIdx);
  }
}
__global__ void s2g_gpu_scatter_kernel(int *in, int *out, int len) {
  int inIdx = blockIdx.x*blockDim.x+threadIdx.x;
  int outIdx = blockIdx.y*blockDim.y+threadIdx.y;
  if(inIdx<len && outIdx < len){
    int intermediate = outInvariant(in[inIdx]);
    atomicAdd(&out[outIdx],outDependent(intermediate, inIdx, outIdx));   
  }
}
static void s2g_cpu_scatter(int *in, int *out, int len) {
  for (int inIdx = 0; inIdx < len; ++inIdx) {
    int intermediate = outInvariant(in[inIdx]);
    for (int outIdx = 0; outIdx < len; ++outIdx) {
      out[outIdx] += outDependent(intermediate, inIdx, outIdx);
    }
  }
}
static void s2g_gpu_scatter(int *in, int *out, int len) {
  printf("start gpu scatter\n");
  int bs = 16;
  dim3 grid((len+bs-1)/bs, (len+bs-1)/bs, 1);
  dim3 block(bs, bs, 1);
  s2g_gpu_scatter_kernel<<<grid, block>>>(in, out, len); 
}

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  int *hostInput;
  int *hostOutput;
  int *deviceInput;
  int *deviceOutput;
  size_t byteCount;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (int *)wbImport(wbArg_getInputFile(args, 0), &inputLength,
                              "Integer");
  hostOutput = (int *)malloc(inputLength * sizeof(int));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);
  byteCount = inputLength * sizeof(int);

  wbTime_start(GPU, "Allocating GPU memory.");
  wbCheck(hipMalloc((void **)&deviceInput, byteCount));
  wbCheck(hipMalloc((void **)&deviceOutput, byteCount));
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  wbCheck(hipMemcpy(deviceInput, hostInput, byteCount,
                     hipMemcpyHostToDevice));
  wbCheck(hipMemset(deviceOutput, 0, byteCount));
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //////////////////////////////////////////
  // GPU Scatter Computation
  //////////////////////////////////////////
  wbTime_start(Compute, "Performing GPU Scatter computation");
  s2g_gpu_scatter(deviceInput, deviceOutput, inputLength);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing GPU Scatter computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  wbCheck(hipMemcpy(hostOutput, deviceOutput, byteCount,
                     hipMemcpyDeviceToHost));
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbSolution(args, hostOutput, inputLength);

  wbTime_start(GPU, "Freeing GPU Memory");
  hipFree(deviceInput);
  hipFree(deviceOutput);
  wbTime_stop(GPU, "Freeing GPU Memory");

  free(hostInput);
  free(hostOutput);

  return 0;
}
