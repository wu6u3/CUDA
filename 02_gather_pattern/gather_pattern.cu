#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

__host__ __device__ int outInvariant(int inValue) {
  return inValue * inValue;
}

__host__ __device__ int outDependent(int value, int inIdx, int outIdx) {
  if (inIdx == outIdx) {
    return 2 * value;
  } else if (inIdx > outIdx) {
    return value / (inIdx - outIdx);
  } else {
    return value / (outIdx - inIdx);
  }
}

__global__ void s2g_gpu_gather_kernel(int *in, int *out, int len) {
  int outIdx = blockIdx.x*blockDim.x+threadIdx.x;
  int results=0;
  if(outIdx < len){
    for (int inIdx = 0; inIdx < len; ++inIdx) {
      int intermediate = outInvariant(in[inIdx]);
      results += outDependent(intermediate, inIdx, outIdx);
    }
    out[outIdx]=results;
  }
}

static void s2g_cpu_scatter(int *in, int *out, int len) {
  for (int outIdx = 0; outIdx < len; ++outIdx) {
    for (int inIdx = 0; inIdx < len; ++inIdx) {
      int intermediate = outInvariant(in[inIdx]);
      out[outIdx] += outDependent(intermediate, inIdx, outIdx);
    }
  }
}

static void s2g_gpu_gather(int *in, int *out, int len) {
 
  printf("start gpu scatter\n");
  int bs = 16;
  
  dim3 grid((len+bs-1)/bs, 1, 1);
  dim3 block(bs, 1, 1);
  
  s2g_gpu_gather_kernel<<<grid, block>>>(in, out, len);
  
}

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  int *hostInput;
  int *hostOutput;
  int *deviceInput;
  int *deviceOutput;
  size_t byteCount;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (int *)wbImport(wbArg_getInputFile(args, 0), &inputLength,
                              "Integer");
  hostOutput = (int *)malloc(inputLength * sizeof(int));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);
  byteCount = inputLength * sizeof(int);

  wbTime_start(GPU, "Allocating GPU memory.");
  wbCheck(hipMalloc((void **)&deviceInput, byteCount));
  wbCheck(hipMalloc((void **)&deviceOutput, byteCount));
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  wbCheck(hipMemcpy(deviceInput, hostInput, byteCount,
                     hipMemcpyHostToDevice));
  wbCheck(hipMemset(deviceOutput, 0, byteCount));
  wbTime_stop(GPU, "Copying input memory to the GPU.");


  //////////////////////////////////////////
  // GPU Gather Computation
  //////////////////////////////////////////
  wbTime_start(Compute, "Performing GPU Gather computation");
  s2g_gpu_gather(deviceInput, deviceOutput, inputLength);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing GPU Gather computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  wbCheck(hipMemcpy(hostOutput, deviceOutput, byteCount,
                     hipMemcpyDeviceToHost));
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbSolution(args, hostOutput, inputLength);

  wbTime_start(GPU, "Freeing GPU Memory");
  hipFree(deviceInput);
  hipFree(deviceOutput);
  wbTime_stop(GPU, "Freeing GPU Memory");

  free(hostInput);
  free(hostOutput);

  return 0;
}
